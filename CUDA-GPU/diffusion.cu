#include "hip/hip_runtime.h"
// diffusion.cu
//
// Program to model 1D heat diffusion equation (CUDA)

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>

#include <hip/hip_runtime.h>


float* diffusion_1D(float *a, float *b, long int size, long int time);
float* diffusion_2D(float *a, float *b, long int size, long int time);


int main (int argc, char *argv[])
{
    float *temp;
    float *array1, *array2;
    long int i, j, rodsize, tempsteps;
    long int index;
    int dim;
    struct timeval start, finish;
    char out_file[100];
    FILE *fp;

    if (argc != 5)
    {
        printf("usage: ./diffusion partitions tempsteps dimensions outfile\n");
        printf("    partitions: number of discrete points along a dimension\n");
        printf("    tempsteps: number of time steps to numerically estimate diffusion\n");
        printf("    dimensions: number of dimensions to perform the diffusion (1 or 2)\n");
        printf("    outfile: name of the output file for the diffusion results\n");
        exit(-1);
    }

    rodsize = atol(argv[1]);
    tempsteps = atol(argv[2]);
    dim = atoi(argv[3]);
    strncpy(out_file, argv[4], 100);

    fp = fopen(out_file, "w");
    if (ferror(fp) || fp == NULL)
    {
        fprintf(stderr, "ERROR: cannot open file %s for printing. Terminating...\n", out_file);
        exit(-1);
    }

    if (dim == 1)
    {
        array1 = (float *) malloc((rodsize + 2) * sizeof (float));
        array2 = (float *) malloc((rodsize + 2) * sizeof (float));

        /* populate the 1D grid with initial temperature values */
        array1[0] = 100.0;
        array2[0] = 100.0;
        for (i = 1; i <= rodsize; i++)
        {
            array1[i] = 23.0;
            array2[i] = 23.0;
        }
        array1[rodsize + 1] = 23.0;
        array2[rodsize + 1] = 23.0;

        /* perform numeric estimation of the 1D diffusion over time */
        gettimeofday(&start, NULL);
        temp = diffusion_1D(array1, array2, rodsize, tempsteps);
        gettimeofday(&finish, NULL);

        /* print diffusion info at a particular point */
        index = (int) (0.7 * rodsize);
        printf("Temperature at location 0.7 at time %ld: %7.2f\n", tempsteps, temp[index]);
        printf("Elapsed time: %f sec.\n\n", (finish.tv_sec - start.tv_sec)
                + (finish.tv_usec - start.tv_usec) / 1000000.0);

        /* write results to file */
        for (i = 1; i <= rodsize; ++i)
        {
            fprintf(fp, "%7.2f\n", temp[i]);
        }

        free(array2);
        free(array1);
    }

    if (dim == 2)
    {
        array1 = (float *) malloc((rodsize + 2) * (rodsize + 2) * sizeof(float));
        array2 = (float *) malloc((rodsize + 2) * (rodsize + 2) * sizeof(float));

        /* populate the 2D grid with initial temperature values */
        for (j = 0; j < rodsize + 2; j++)
        {
            array1[j] = 100.0;
            array2[j] = 100.0;
        }
        for (i = 1; i < rodsize + 2; i++)
        {
            for (j = 0; j < rodsize + 2; j++)
            {
                array1[i * (rodsize + 2) + j] = 23.0;
                array2[i * (rodsize + 2) + j] = 23.0;
            }
        }

        /* perform numeric estimation of the 2D diffusion over time */
        gettimeofday(&start, NULL);
        temp = diffusion_2D(array1, array2, rodsize, tempsteps);
        gettimeofday(&finish, NULL);

        /* print diffusion info at a particular point */
        index = (int) (0.7 * rodsize);
        printf("Temperature at location 0.7 at time %ld: %7.2f\n", tempsteps, temp[index]);
        printf("Elapsed time: %f sec.\n\n", (finish.tv_sec - start.tv_sec)
                + (finish.tv_usec - start.tv_usec) / 1000000.0);

        for (i = 1; i <= rodsize; ++i)
        {
            for (j = 1; j <= rodsize; ++j)
            {
                fprintf(fp, "%7.2f ", temp[i * (rodsize + 2) + j]);
            }
            fprintf(fp, "\n");
        }

        free(array2);
        free(array1);
    }

    fclose(fp);

    return 0;
}


__global__ void diffusion_1D_kernel(float *a, float *b, long int size)
{

    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k != 0  && k < size)
        a[k] = (b[k - 1] + b[k + 1]) / 2.0;
    
    if (k==0)
        a[size + 1] = a[size];
}


/* Compute the diffusion over a 1D grid over the prescribed time
 *
 * a/b: arrays containing the current and next temperatures over the 1D grid
 *  note: b must contain the initial grid values
 * size: total number of elements in the 2D grid
 * time: number of time steps for which to diffuse (arbitrary units)
 */
float* diffusion_1D(float *a, float *b, long int size, long int time)
{

    hipError_t err;
    
    int siz = (size + 2) * sizeof(float);     
    float *d_A, *d_B;    
    float *temp = (float*) malloc(siz);
    
    err = hipMalloc((void **) &d_A, siz);
    if (err){
        printf("Error = %s\n", hipGetErrorString(err));
    }
    
    
    
    hipMemcpy(d_A, a, siz, hipMemcpyHostToDevice);

      

      
    hipMalloc((void **) &d_B, siz);
    
    hipMemcpy(d_B, b, siz, hipMemcpyHostToDevice);
    if (err)
        printf("Error =  %s\n", hipGetErrorString(err));

    int numBlocks = ceil(size / 256.0);
    int i = 0;
    float *tempPtr;
    dim3 threadsPerBlock(size);    
    for (i=0; i<time; i++){
        diffusion_1D_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, size);
        
        tempPtr = d_A;
        d_A = d_B;
        d_B = tempPtr;
        
    }
    

    hipDeviceSynchronize();
    
    
    hipMemcpy(a, d_A, size, hipMemcpyDeviceToHost);   

    hipFree(d_A); 
    hipFree(d_B);
    
    return a;
}


__global__ void diffusion_2D_kernel(float *a, float *b, long int size)
{

    long int i, j, k, siz;

    siz = (size + 2);
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i != 0  && i < size)
        for(j = 1; j <= size; j++)
            for (k = 1; k <= size; k++)
                a[j*siz + k] = (b[j*siz + k-1] + b[j*siz + k + 1] + b[(j-1) * siz + k] + b[(j+1)*siz +k]) / 4.0;
            
        
    

    if(i == 0)
        a[(size+1)*siz] = a[size*siz];


}


/* Compute the diffusion over a 2D grid over the prescribed time
 *
 * a/b: arrays containing the current and next temperatures over the 2D grid
 *  note: arrays will be linearized (flattened) from 2D to 1D
 *  note: b must contain the initial grid values
 * size: total number of elements in the 2D grid
 * time: number of time steps for which to diffuse (arbitrary units)
 */
float* diffusion_2D(float *a, float *b, long int size, long int time){

    float *d_A, *d_B; 
    
    int siz = (size + 2)* (size+2) * sizeof(float);         
    float *temp = (float*) malloc(siz);
    
    hipMalloc((void **) &d_A, siz);

    hipMemcpy(d_A, a, siz, hipMemcpyHostToDevice);
      
    hipMalloc((void **) &d_B, siz);
    
    hipMemcpy(d_B, b, siz, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(size);
    int numBlocks = ceil(size / 256.0);
    int r = 0;
    for(r = 0; r < time; r++)
    {
        /* compute next grid values (a = next, b = current) */
        diffusion_2D_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, size);

        hipDeviceSynchronize();
        
        /* swap current and new grid points */
        temp = d_A;
        d_A = d_B;
        d_B = temp;
    }


    hipDeviceSynchronize();
    
    hipMemcpy(a, d_A, siz, hipMemcpyDeviceToHost);   

    hipFree(d_A); 
    hipFree(d_B);

    return a;
}
